#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "blur_device.cuh"


__global__
void cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    const unsigned int n_frames, const unsigned int blur_v_size) {

    // TODO: Fill in the implementation for the GPU-accelerated convolution. 
    //
    // It may be helpful to use the information in the lecture slides, as well
    // as the CPU implementation, as a reference.
    
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("vsize: %d\n", &blur_v_size);
    while (thread_index < n_frames) {
        if (thread_index < blur_v_size) {
            for (unsigned int i = 0; i <= thread_index; ++i) {
                out_data[thread_index] += raw_data[thread_index - i] * blur_v[i];
            }
        } else {
            for (unsigned int j = 0; j < blur_v_size; ++j) {
                out_data[thread_index] += raw_data[thread_index - j] * blur_v[j];
            }
        }
        thread_index += blockDim.x * gridDim.x;
        
        //printf("b: %d\n", &blockIdx.x);
        //printf("g: %d\n", &gridDim.x);
    }
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int n_frames,
        const unsigned int blur_v_size) {
        
    // Call the kernel above this function.
    cudaBlurKernel<<<blocks, threadsPerBlock>>>(raw_data, blur_v, out_data, n_frames, blur_v_size);
}
