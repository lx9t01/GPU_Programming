#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "blur_device.cuh"


__global__
void cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    int n_frames, int blur_v_size) {

    // TODO: Fill in the implementation for the GPU-accelerated convolution. 
    //
    // It may be helpful to use the information in the lecture slides, as well
    // as the CPU implementation, as a reference.
    
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (thread_index < n_frames) {
        if (thread_index < (unsigned int)blur_v_size) {
            for (int i = 0; i <= thread_index; i += blockDim.x * gridDim.x) {
                out_data[thread_index] += raw_data[thread_index - i] * blur_v[i];
            }
        } else {
            for (int i = 0; i < (unsigned int)blur_v_size; i += blockDim.x * gridDim.x) {
                out_data[thread_index] += raw_data[thread_index - i] * blur_v[i];
            }
        }
        thread_index += blockDim.x * gridDim.x;
    }
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int n_frames,
        const unsigned int blur_v_size) {
        
    // Call the kernel above this function.
    cudaBlurKernel<<<blocks, threadsPerBlock>>>(raw_data, blur_v, out_data, n_frames, blur_v_size);
}
