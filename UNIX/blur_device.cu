#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "blur_device.cuh"


__global__
void cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    unsigned int n_frames, unsigned int blur_v_size) {

    // TODO: Fill in the implementation for the GPU-accelerated convolution. 
    //
    // It may be helpful to use the information in the lecture slides, as well
    // as the CPU implementation, as a reference.
    
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (thread_index < n_frames) {
        if (thread_index < blur_v_size) {
            for (unsigned int i = 0; i <= thread_index; ++i) {
                out_data[thread_index] += raw_data[thread_index - i] * blur_v[i];
            }
        } else {
            for (unsigned int i = 0; i < blur_v_size; ++i) {
                out_data[thread_index] += raw_data[thread_index - i] * blur_v[i];
            }
        }
        thread_index += blockDim.x * gridDim.x;
        printf("t: %d\n", &thread_index);
        //printf("b: %d\n", &blockIdx.x);
        //printf("g: %d\n", &gridDim.x);
    }
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int n_frames,
        const unsigned int blur_v_size) {
        
    // Call the kernel above this function.
    cudaBlurKernel<<<blocks, threadsPerBlock>>>(raw_data, blur_v, out_data, n_frames, blur_v_size);
}
